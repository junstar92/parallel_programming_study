#include "hip/hip_runtime.h"
/*****************************************************************************
 * File:        smemRectangular.cu
 * Description: This is an example of using shared memory to transpose rectangular
 *              thread coordinates of a CUDA grid into a global memory array.
 *              Different kernels below demonstrae performing reads and writes
 *              with different ordering, as optimizing using memory padding.
 *              
 * Compile:     nvcc -o smemRectangular smemRectangular.cu -I..
 * Run:         ./smemRectangular
 *****************************************************************************/
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "common/common.h"

#define BDIMX 32
#define BDIMY 16
#define IPAD 2

__global__
void setRowReadRow(int* out)
{
    // static shared memory
    __shared__ int tile[BDIMY][BDIMX];

    // mapping from thread index to global memory index
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    // shared memory store operation
    tile[threadIdx.y][threadIdx.x] = idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[idx] = tile[threadIdx.y][threadIdx.x];
}

__global__
void setColReadCol(int* out)
{
    // static shared memory
    __shared__ int tile[BDIMX][BDIMY];

    // mapping from thread index to global memory index
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    // shared memory store operation
    tile[threadIdx.x][threadIdx.y] = idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[idx] = tile[threadIdx.x][threadIdx.y];
}

__global__
void setColReadCol2(int *out)
{
    // static shared memory
    __shared__ int tile[BDIMY][BDIMX];

    // mapping from 2D thread index to linear memory
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    // convert idx to transposed coordinate (row, col)
    unsigned int irow = idx / blockDim.y;
    unsigned int icol = idx % blockDim.y;

    // shared memory store operation
    tile[icol][irow] = idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[idx] = tile[icol][irow] ;
}

__global__
void setRowReadCol(int *out)
{
    // static shared memory
    __shared__ int tile[BDIMY][BDIMX];

    // mapping from 2D thread index to linear memory
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    // convert idx to transposed coordinate (row, col)
    unsigned int irow = idx / blockDim.y;
    unsigned int icol = idx % blockDim.y;

    // shared memory store operation
    tile[threadIdx.y][threadIdx.x] = idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[idx] = tile[icol][irow];
}

__global__
void setRowReadColPad(int *out)
{
    // static shared memory
    __shared__ int tile[BDIMY][BDIMX + IPAD];

    // mapping from 2D thread index to linear memory
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    // convert idx to transposed (row, col)
    unsigned int irow = idx / blockDim.y;
    unsigned int icol = idx % blockDim.y;

    // shared memory store operation
    tile[threadIdx.y][threadIdx.x] = idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[idx] = tile[icol][irow];
}

__global__
void setRowReadColDyn(int *out)
{
    // dynamic shared memory
    extern  __shared__ int tile[];

    // mapping from thread index to global memory index
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

    // convert idx to transposed (row, col)
    unsigned int irow = idx / blockDim.y;
    unsigned int icol = idx % blockDim.y;

    // convert back to smem idx to access the transposed element
    unsigned int col_idx = icol * blockDim.x + irow;

    // shared memory store operation
    tile[idx] = idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[idx] = tile[col_idx];
}

__global__
void setRowReadColDynPad(int *out)
{
    // dynamic shared memory
    extern  __shared__ int tile[];

    // mapping from thread index to global memory index
    unsigned int g_idx = threadIdx.y * blockDim.x + threadIdx.x;

    // convert idx to transposed (row, col)
    unsigned int irow = g_idx / blockDim.y;
    unsigned int icol = g_idx % blockDim.y;

    unsigned int row_idx = threadIdx.y * (blockDim.x + IPAD) + threadIdx.x;

    // convert back to smem idx to access the transposed element
    unsigned int col_idx = icol * (blockDim.x + IPAD) + irow;

    // shared memory store operation
    tile[row_idx] = g_idx;

    // wait for all threads to complete
    __syncthreads();

    // shared memory load operation
    out[g_idx] = tile[col_idx];
}

int main(int argc, char** argv)
{
    // setup device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CUDA_CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Starting at device %d: %s\n", dev, deviceProp.name);
    CUDA_CHECK(hipSetDevice(dev));

    hipSharedMemConfig pConfig;
    CUDA_CHECK(hipDeviceGetSharedMemConfig ( &pConfig ));
    printf("with Bank Mode:%s \n", pConfig == 1 ? "4-Byte" : "8-Byte");

    // set up array size 2048
    int nx = BDIMX;
    int ny = BDIMY;

    size_t nBytes = nx * ny * sizeof(int);

    // execution configuration
    dim3 block (BDIMX, BDIMY);
    dim3 grid  (1, 1);
    printf("<<< grid (%d,%d) block (%d,%d)>>>\n", grid.x, grid.y, block.x, block.y);

    // allocate device memory
    int *d_C;
    CUDA_CHECK(hipMalloc((void**)&d_C, nBytes));
    int *gpuRef  = (int *)malloc(nBytes);

    CUDA_CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadRow<<<grid, block>>>(d_C);
    CUDA_CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipMemset(d_C, 0, nBytes));
    setColReadCol<<<grid, block>>>(d_C);
    CUDA_CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipMemset(d_C, 0, nBytes));
    setColReadCol2<<<grid, block>>>(d_C);
    CUDA_CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadCol<<<grid, block>>>(d_C);
    CUDA_CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadColDyn<<<grid, block, BDIMX*BDIMY*sizeof(int)>>>(d_C);
    CUDA_CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadColPad<<<grid, block>>>(d_C);
    CUDA_CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadColDynPad<<<grid, block, (BDIMX + IPAD)*BDIMY*sizeof(int)>>>(d_C);
    CUDA_CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    // free host and device memory
    CUDA_CHECK(hipFree(d_C));
    free(gpuRef);

    // reset device
    CUDA_CHECK(hipDeviceReset());
    return 0;
}