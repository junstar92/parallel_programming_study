// 02_first-parallel.cu

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void firstParallel()
{
    printf("This is running in parallel.\n");
}

int main()
{
    firstParallel<<<5, 5>>>();
    hipDeviceSynchronize();
}