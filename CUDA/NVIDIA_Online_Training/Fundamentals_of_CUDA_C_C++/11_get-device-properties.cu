// 11_get-device-properties

#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
    /*
     * Device ID is required first to query the device.
     */

    int deviceId;
    hipGetDevice(&deviceId);

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceId);

    /*
     * `props` now contains several properties about the current device.
     */

    int computeCapabilityMajor = props.major;
    int computeCapabilityMinor = props.minor;
    int multiProcessorCount = props.multiProcessorCount;
    int warpSize = props.warpSize;

    printf("Device ID: %d\nNumber of SMs: %d\nCompute Capability Major: %d\nCompute Capability Minor: %d\nWarp Size: %d\n", deviceId, multiProcessorCount, computeCapabilityMajor, computeCapabilityMinor, warpSize);
}