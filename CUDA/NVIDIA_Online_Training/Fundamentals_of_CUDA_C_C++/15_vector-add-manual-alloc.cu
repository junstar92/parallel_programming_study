// 15_vector-add-manual-alloc.cu

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

__global__
void initWith(float num, float *a, int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < N; i += stride)
    {
        a[i] = num;
    }
}

__global__
void addVectorsInto(float* result, float* a, float* b, const int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < N; i += stride) {
        result[i] = a[i] + b[i];
    }
}

void checkElementsAre(float target, float* array, const int N)
{
    for (int i = 0; i < N; i++) {
        if (array[i] != target) {
            printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
            exit(1);
        }
    }
    printf("SUCCESS! All values added correctly.\n");
}

int main()
{
    const int N = 2 << 20;
    size_t size = N * sizeof(float);

    int deviceId;
    checkCuda(hipGetDevice(&deviceId));

    hipDeviceProp_t props;
    checkCuda(hipGetDeviceProperties(&props, deviceId));

    float *a, *b, *c, *h_c;

    checkCuda(hipMalloc(&a, size));
    checkCuda(hipMalloc(&b, size));
    checkCuda(hipMalloc(&c, size));
    checkCuda(hipHostMalloc(&h_c, size, hipHostMallocDefault));

    size_t threadsPerBlock = props.maxThreadsPerBlock;
    size_t numberOfBlocks = props.multiProcessorCount;

    hipStream_t stream1, stream2, stream3;
    checkCuda(hipStreamCreate(&stream1));
    checkCuda(hipStreamCreate(&stream2));
    checkCuda(hipStreamCreate(&stream3));

    initWith<<<numberOfBlocks, threadsPerBlock, 0, stream1>>>(3, a, N);
    initWith<<<numberOfBlocks, threadsPerBlock, 0, stream2>>>(4, b, N);
    initWith<<<numberOfBlocks, threadsPerBlock, 0, stream3>>>(0, c, N);

    addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

    checkCuda(hipMemcpy(h_c, c, size, hipMemcpyDeviceToHost));
    
    checkElementsAre(7, h_c, N);

    checkCuda(hipStreamDestroy(stream1));
    checkCuda(hipStreamDestroy(stream2));
    checkCuda(hipStreamDestroy(stream3));

    checkCuda(hipFree(a));
    checkCuda(hipFree(b));
    checkCuda(hipFree(c));
    checkCuda(hipHostFree(h_c));
}