// 13_print-numbers.cu

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void printNumber(int number)
{
    printf("%d\n", number);
}

int main()
{
    for (int i = 0; i < 5; i++) {
        hipStream_t stream;
        hipStreamCreate(&stream);
        printNumber<<<1, 1, 0, stream>>>(i);
        hipStreamDestroy(stream);
    }

    hipDeviceSynchronize();
}