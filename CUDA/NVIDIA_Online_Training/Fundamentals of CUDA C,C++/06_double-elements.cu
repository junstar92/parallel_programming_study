#include "hip/hip_runtime.h"
// 06_double-elements.cu
#include <stdio.h>

void init(int *a, const int N)
{
    for (int i = 0; i < N; i++) {
        a[i] = i;
    }
}

__global__
void doubleElements(int *a, const int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        a[i] *= 2;
    }
}

bool checkElementsAreDoubled(int *a, const int N)
{
    for (int i = 0; i < N; i++) {
        if (a[i] != i * 2)
            return false;
    }

    return true;
}

int main()
{
    int N = 1000;
    int *a;

    size_t size = N * sizeof(int);

    // Use 'hipMallocManaged' to allocate pointer 'a' available
    // on both the host and the device.
    cudamallocManaged(&a, size);

    init(a, N);

    size_t threads_per_block = 256;
    size_t number_of_blocks = (N + threads_per_block - 1) / threads_per_block;

    doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
    hipDeviceSynchronize();
    
    bool areDoubled = checkElementsAreDoubled(a, N);
    printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

    // Use 'hipFree' to free memory allocated with 'hipMallocManaged'
    hipFree(a);
}