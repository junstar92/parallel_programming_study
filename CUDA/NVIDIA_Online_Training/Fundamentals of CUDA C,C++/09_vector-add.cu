// 09_vector-add.cu

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

void initWith(float num, float* a, const int N)
{
    for (int i = 0; i < N; i++) {
        a[i] = num;
    }
}

__global__
void addVectorsInto(float* result, float* a, float* b, const int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < N; i += stride) {
        result[i] = a[i] + b[i];
    }
}

void checkElementsAre(float target, float* array, const int N)
{
    for (int i = 0; i < N; i++) {
        if (array[i] != target) {
            printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
            exit(1);
        }
    }
    printf("SUCCESS! All values added correctly.\n");
}

int main()
{
    const int N = 2 << 20;
    size_t size = N * sizeof(float);

    float *a, *b, *c;

    checkCuda(hipMallocManaged(&a, size));
    checkCuda(hipMallocManaged(&b, size));
    checkCuda(hipMallocManaged(&c, size));

    initWith(3, a, N);
    initWith(4, b, N);
    initWith(0, c, N);

    size_t threadsPerBlock = 256;
    size_t numberOfBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

    checkCuda(hipGetLastError());
    checkCuda(hipDeviceSynchronize());

    checkElementsAre(7, c, N);

    checkCuda(hipFree(a));
    checkCuda(hipFree(b));
    checkCuda(hipFree(c));
}