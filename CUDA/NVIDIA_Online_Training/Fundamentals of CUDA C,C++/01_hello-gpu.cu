
#include <hip/hip_runtime.h>
#include <stdio.h>

void helloCPU()
{
    printf("Hello from the CPU.\n");
}

__global__ void helloGPU()
{
    printf("Hello from the GPU.\n");
}

int main()
{
    helloCPU();

    helloGPU<<<1, 1>>>();
    hipDeviceSynchronize();
}