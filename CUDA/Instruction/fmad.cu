#include "hip/hip_runtime.h"
/*****************************************************************************
 * File:        fmad.cu
 * Description: This is an example to illustrates the effect on numerical accuracy
 *              of fusing a multiply-add into a sing MAD instruction.
 *              
 * Compile:     nvcc -o fmad fmad.cu -I..
 * Run:         ./fmad
 *****************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "common/common.h"

__global__
void fmad_kernel(double x, double y, double *out)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid == 0) {
        *out = x * x + y;
    }
}

double host_fmad_kernel(double x, double y)
{
    return x * x + y;
}

int main(int argc, char** argv)
{
    double *d_out, h_out;
    double x = 2.891903;
    double y = -3.980364;

    double host_value = host_fmad_kernel(x, y);

    CUDA_CHECK(hipMalloc(void**)&d_out, sizeof(double));
    fmad_kernel<<<1, 32>>>(x, y, d_out);
    CUDA_CHECK(hipMemcpy(&h_out, d_out, sizeof(double), hipMemcpyDeviceToHost));

    if (host_value == h_out) {
        printf("The device output the same value as the host.\n");
    }
    else {
        printf("The device output a different value than the host, diff=%e.\n", fabs(host_value - h_out));
    }

    return 0;
}