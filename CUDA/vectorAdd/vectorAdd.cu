#include "hip/hip_runtime.h"
/*****************************************************************************
 * File:        vectorAdd.cu
 * Description: Vector addition, C = A + B
 *              This code is a basic sample that implements element by element
 *              vector addition.
 *              
 * Compile:     nvcc -o vectorAdd vectorAdd.cu -I.. -lcuda
 * Run:         ./vectorAdd <n>
 *                  <n> : the number of elements in vector
 *****************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <common/common.h>

void Usage(char prog_name[]);
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements);

int main(int argc, char* argv[])
{
    if (argc != 2) {
        Usage(argv[0]);
    }

    int numElements = strtol(argv[1], NULL, 10);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vectors A, B, C
    float *h_A = (float*)malloc(numElements * sizeof(float));
    float *h_B = (float*)malloc(numElements * sizeof(float));
    float *h_C = (float*)malloc(numElements * sizeof(float));

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_SUCCESS);
    }

    // Initialize that host input vectors
    common_init_rand_fvec(h_A, numElements);
    common_init_rand_fvec(h_B, numElements);

    // Allocate the device input vectors A, B, C
    float *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc((void**)&d_A, numElements * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_B, numElements * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_C, numElements * sizeof(float)));

    // Copy the host input vector A and B in host memory 
    // to the device input vectors in device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    CUDA_CHECK(hipMemcpy(d_A, h_A, numElements * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, numElements * sizeof(float), hipMemcpyHostToDevice));

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    double start, finish;
    GET_TIME(start);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    CUDA_CHECK(hipDeviceSynchronize());
    GET_TIME(finish);
    CUDA_CHECK(hipGetLastError());

    // Copy the device result vector in device memory
    // to the host result vector in host memory
    printf("Copy output data from the CUDA device to the host memory\n");
    CUDA_CHECK(hipMemcpy(h_C, d_C, numElements * sizeof(float), hipMemcpyDeviceToHost));

    // Verify that the result vector is correct
    printf("Random Sampling Verifying...\n");
    for (int i = 0; i < 10; i++) {
        int idx = rand() % numElements;
        printf("[INDEX %d] %f + %f = %f\n", idx, h_A[idx], h_B[idx], h_C[idx]);
        if (fabs(h_A[idx] + h_B[idx] - h_C[idx]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d\n", idx);
            exit(EXIT_FAILURE);
        }
    }
    printf(".....\n");
    printf("Test PASSED\n");
    
    // Free device global memory
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    printf("Elapsed time of kernel function : %f seconds\n", finish-start);
    return 0;
}

void Usage(char prog_name[])
{
    fprintf(stderr, "Usage: %s <n>\n", prog_name);
    fprintf(stderr, "\t<n> : the number of elements in vector\n");
    exit(EXIT_FAILURE);
}

__global__
void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < numElements)
        C[i] = A[i] + B[i];
}