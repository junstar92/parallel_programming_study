#include "hip/hip_runtime.h"
/*****************************************************************************
 * File:        simpleHyperQDependece.cu
 * Description: This is an example of adding inter-stream dependencies using
 *              hipStreamWaitEvent. This code launches 4 kernels in each of
 *              n_streams streams. An event is recoded at the completion of
 *              each stream (kernelEvent). cudaStreamEvent is then called on
 *              that event and the last stream to force all computation in the
 *              final stream to only execute when all other streams have completed.
 *              
 * Compile:     nvcc -o simpleHyperQDependece simpleHyperQDependece.cu -I..
 * Run:         ./simpleHyperQDependece
 *****************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "common/common.h"

#define N 1000
#define NSTREAM 4

__global__
void kernel_1()
{
    double sum = 0.0;

    for (int i = 0; i < N; i++) {
        sum = sum + tan(0.1) * tan(0.1);
        printf("%f\n", sum);
    }
}

__global__
void kernel_2()
{
    double sum = 0.0;

    for (int i = 0; i < N; i++) {
        sum = sum + tan(0.1) * tan(0.1);
        printf("%f\n", sum);
    }
}

__global__
void kernel_3()
{
    double sum = 0.0;

    for (int i = 0; i < N; i++) {
        sum = sum + tan(0.1) * tan(0.1);
        printf("%f\n", sum);
    }
}

__global__
void kernel_4()
{
    double sum = 0.0;

    for (int i = 0; i < N; i++) {
        sum = sum + tan(0.1) * tan(0.1);
        printf("%f\n", sum);
    }
}

int main(int argc, char** argv)
{
    int n_streams = NSTREAM;
    int isize = 1;
    int iblock = 1;
    int bigcase = 0;

    // get argument from command line
    if (argc > 1)
        n_streams = atoi(argv[1]);
    if (argc > 2)
        bigcase = atoi(argv[2]);
    
    float elapsed_time;

    // set up max connection
    char* iname = "CUDA_DEVICE_MAX_CONNECTIONS";
    _putenv_s(iname, "32");
    char* ivalue = getenv(iname);
    printf("%s = %s\n", iname, ivalue);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CUDA_CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("> Using Device %d: %s with num_streams=%d\n", dev, deviceProp.name, n_streams);
    CUDA_CHECK(hipSetDevice(dev));

    // check if device support hyper-Q
    if (deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5)) {
        if (deviceProp.concurrentKernels == 0) {
            printf("> GPU does not support concurrent kernel execution (SM 3.5 or higher required)\n");
            printf("> CUDA kernel runs will be serialized\n");
        }
        else {
            printf("> GPU does not support HyperQ\n");
            printf("> CUDA kernel runs will have limited concurrency\n");
        }
    }

    printf("> Compute Capability %d.%d hardware with %d multi-processors\n", deviceProp.major,
            deviceProp.minor, deviceProp.multiProcessorCount);
    
    // Allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t*)malloc(n_streams * sizeof(hipStream_t));

    for (int i = 0; i < n_streams; i++) {
        CUDA_CHECK(hipStreamCreate(&(streams[i])));
    }
    
    // run kernel with more threads
    if (bigcase == 1) {
        iblock = 512;
        isize = 1 << 12;
    }

    // setup execution configuration
    dim3 block(iblock);
    dim3 grid(isize / iblock);
    printf("> grid %d block %d\n", grid.x, block.x);

    // create events
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    hipEvent_t *kernelEvent;
    kernelEvent = (hipEvent_t*)malloc(n_streams * sizeof(hipEvent_t));
    for (int i = 0; i < n_streams; i++) {
        CUDA_CHECK(hipEventCreateWithFlags(&(kernelEvent[i]), hipEventDisableTiming));
    }

    // record start event
    CUDA_CHECK(hipEventRecord(start, 0));

    // dispatch job with depth first ordering
    for (int i = 0;i < n_streams; i++) {
        kernel_1<<<grid, block, 0, streams[i]>>>();
        kernel_2<<<grid, block, 0, streams[i]>>>();
        kernel_3<<<grid, block, 0, streams[i]>>>();
        kernel_4<<<grid, block, 0, streams[i]>>>();

        CUDA_CHECK(hipEventRecord(kernelEvent[i], streams[i]));
        CUDA_CHECK(hipStreamWaitEvent(streams[n_streams-1], kernelEvent[i], 0));
    }
    
    // record stop event
    CUDA_CHECK(hipEventRecord(stop, 0));
    CUDA_CHECK(hipEventSynchronize(stop));

    // calculate elapsed time
    CUDA_CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("Measured time for parallel execution = %fs\n", elapsed_time / 1000.f);

    // release all streams
    for (int i = 0; i < n_streams; i++) {
        CUDA_CHECK(hipStreamDestroy(streams[i]));
        CUDA_CHECK(hipEventDestroy(kernelEvent[i]));
    }
    free(streams);
    free(kernelEvent);
    
    // destory events
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    // reset device
    CUDA_CHECK(hipDeviceReset());

    return 0;
}