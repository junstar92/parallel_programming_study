#include "hip/hip_runtime.h"
/*****************************************************************************
 * File:        matrixMulTiling.cu
 * Description: Matrix multiplication by using tiling, C = AB
 *              A has m x k dimensions, B has k x n dimensions, and C has
 *              m x n dimensions.
 *              It is not for the most performance.
 *              
 * Compile:     nvcc -o matrixMulTiling matrixMulTiling.cu -I.. -lcuda
 * Run:         ./matrixMulTiling <m> <k> <n>
 *                  <m> : the number of rows in Matrix A
 *                  <k> : the number of columns in Matrix A, it is also
 *                        the number of rows in Matrix B.
 *                  <n> : the number of columns in Matrix B.
 *****************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <common/common.h>

#define TILE_WIDTH 32

void Usage(char prog_name[]);
__global__ void matrixMul(const float *A, const float *B, float *C, const int M, const int K, const int N);

int main(int argc, char* argv[])
{
    if (argc != 4) {
        Usage(argv[0]);
    }

    int m = strtol(argv[1], NULL, 10);
    int k = strtol(argv[2], NULL, 10);
    int n = strtol(argv[3], NULL, 10);
    printf("[Matrix multiplication, C = AB]\n");
    printf("\tA is (%d x %d) matrix, B is (%d x %d) matrix, and \n", m, k, k, n);
    printf("\tC is (%d x %d) matrix.\n", m, n);

    // Allocate the host matrix A, B, C
    float *h_A = (float*)malloc(m * k * sizeof(float));
    float *h_B = (float*)malloc(k * n * sizeof(float));
    float *h_C = (float*)malloc(m * n * sizeof(float));

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        fprintf(stderr, "Failed to allocate host matrix!\n");
        exit(EXIT_SUCCESS);
    }
    
    // Initialize that host matrix
    common_random_init_matrix<float>(h_A, m, k);
    common_random_init_matrix<float>(h_B, k, n);

    // Allocate the device matrix A, B, C
    float *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc((void**)&d_A, m * k * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_B, k * n * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_C, m * n * sizeof(float)));

    // Copy the host input matrix A and B in host memory 
    // to the device input matrix in device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    CUDA_CHECK(hipMemcpy(d_A, h_A, m * k * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, k * n * sizeof(float), hipMemcpyHostToDevice));

    // Allocate CUDA events for estimating
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Launch the Matrix Multiplication CUDA Kernel
    const int block_size = TILE_WIDTH;
    dim3 threads(block_size, block_size);
    dim3 grid(ceil(m / (float)threads.x), ceil(n / (float)threads.y));
    printf("CUDA kernel launch with (%d x %d) blocks of (%d x %d) threads\n", grid.x, grid.y, threads.x, threads.y);

    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipEventRecord(start));

    matrixMul<<<grid, threads>>>(d_A, d_B, d_C, m, k, n);

    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipEventRecord(stop));


    // Copy the device result matrix in device memory
    // to the host result matrix in host memory
    printf("Copy output data from the CUDA device to the host memory\n");
    CUDA_CHECK(hipMemcpy(h_C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost));

    // Verify that the result matrix is correct (L2-norm error)
    common_verify_matMul_l2ne(h_A, h_B, h_C, m, k, n);
    
    // Compute and Print the performance
    COMPUTE_MATMUL_PERFORMANCE(start, stop, m, k, n, threads.x * threads.y);
    
    // Free device global memory
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");

    return 0;
}

void Usage(char prog_name[])
{
    fprintf(stderr, "Usage: %s <m> <k> <n>\n", prog_name);
    fprintf(stderr, "\t<m> : the number of rows in matrix A.\n");
    fprintf(stderr, "\t<k> : the number of columns in Matrix A, it is also\n");
    fprintf(stderr, "\t      the number of rows in Matrix B.\n");
    fprintf(stderr, "\t<n> : the number of columns in matrix B.\n");
    exit(EXIT_FAILURE);
}

__global__
void matrixMul(const float *A, const float *B, float *C, const int M, const int K, const int N)
{
    __shared__ float Asub[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bsub[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    float Pvalue = 0;
    for (int h = 0; h < ceil(K / (float)TILE_WIDTH); ++h) {
        if ((Row < M) && (h*TILE_WIDTH + tx < K))
            Asub[ty][tx] = A[Row*K + h*TILE_WIDTH + tx];
        else
            Asub[ty][tx] = 0;
        if ((Col < N) && (h*TILE_WIDTH + ty < K))
            Bsub[ty][tx] = B[(h*TILE_WIDTH + ty)*K + Col];
        else
            Bsub[ty][tx] = 0;

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; k++) {
            Pvalue += Asub[ty][k] * Bsub[k][tx];
        }

        __syncthreads();
    }

    if ((Row < M) && (Col < N))
        C[Row*K + Col] = Pvalue;
}